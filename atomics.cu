
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square( int * d_in){
    int totalSum;
	if (threadIdx.x == 0) totalSum = 0;
    __syncthreads();

    int localVal = d_in[threadIdx.x];
    atomicAdd(&totalSum, 1);
    __syncthreads();
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	int h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = i;
	}
	int * d_in;
	hipMalloc((void**) &d_in, ARRAY_BYTES);
//	cudaMalloc((void*) &totalSum, sizeof(float));
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	square<<<1, ARRAY_SIZE>>>(d_in);
//	cudaMemcpy(ans, totalSum, sizeof(float), cudaMemcpyDeviceToHost);
//    printf("%f\n",ans);
	hipFree(d_in);

	return 0;
}
