
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_out, float * d_in){
	
	int idx = threadIdx.x;
	float f = d_in[idx];
	switch(idx%32){
	    case 0: d_out[idx] = f*f;break;
	    case 1: d_out[idx] = f*f;break;
	    case 2: d_out[idx] = f*f;break;
	    case 3: d_out[idx] = f*f;break;
	    case 4: d_out[idx] = f*f;break;
	    case 5: d_out[idx] = f*f;break;
	    case 6: d_out[idx] = f*f;break;
	    case 7: d_out[idx] = f*f;break;
	    case 8: d_out[idx] = f*f;break;
	    case 9: d_out[idx] = f*f;break;
	    case 10: d_out[idx] = f*f;break;
	    case 11: d_out[idx] = f*f;break;
	    case 12: d_out[idx] = f*f;break;
	    case 13: d_out[idx] = f*f;break;
	    case 14: d_out[idx] = f*f;break;
	    case 15: d_out[idx] = f*f;break;
	    case 16: d_out[idx] = f*f;break;
	    case 17: d_out[idx] = f*f;break;
	    case 18: d_out[idx] = f*f;break;
	    case 19: d_out[idx] = f*f;break;
	    case 20: d_out[idx] = f*f;break;
	    case 21: d_out[idx] = f*f;break;
	    case 22: d_out[idx] = f*f;break;
	    case 23: d_out[idx] = f*f;break;
	    case 24: d_out[idx] = f*f;break;
	    case 25: d_out[idx] = f*f;break;
	    case 26: d_out[idx] = f*f;break;
	    case 27: d_out[idx] = f*f;break;
	    case 28: d_out[idx] = f*f;break;
	    case 29: d_out[idx] = f*f;break;
	    case 30: d_out[idx] = f*f;break;
	    case 31: d_out[idx] = f*f;break;
	}
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	square<<<1, ARRAY_SIZE>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
