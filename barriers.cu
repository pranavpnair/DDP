#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
 unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    vector[id] = id;
    __syncthreads();//barrier here
 
 if(id< vectorsize-1 && vector[id+1]!=id+1)
    printf("Incorrect\n");
    
}
#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned N = 1024;
    unsigned *vector, *hvector;
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
    printf("nblocks = %d\n", nblocks);
    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }
    return 0;
}
