#include<stdio.h>
#include<hip/hip_runtime.h>
#define height 4
#define width 4

// Device code
__global__ void kernel(int* d_A, int pitch)
{
    for (int c = 0; c < height; ++c) {
        for (int r = 0; r < width; ++r) {
             int* row = (int*)((char*)d_A + r * pitch);
             row[c] = row[c]*row[c];
        }
    }
}

//Host Code
int main()
{
    int* d_A;
    size_t pitch;
    int *A;
    int rows = height;
    int cols = width;
    A = (int *)malloc(rows*cols*sizeof(int));
    for (int i = 0; i < rows*cols; i++) A[i] = i;
    hipMallocPitch((void**)&d_A, &pitch, width * sizeof(int), height);
    hipMemcpy2D(d_A, pitch, A, sizeof(int)*cols, sizeof(int)*cols, rows, hipMemcpyHostToDevice);
    kernel<<<100, 32>>>(d_A, pitch);
    hipDeviceSynchronize();
    for(int i=0;i<rows*cols;i++)
        printf("%d %d\n",A[i],d_A[i]);
    return 0;
}



