#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize,int N) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]+=N;  
}

#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned long long N = 1024;
    unsigned *vector, *hvector;
    unsigned vec[N];
    for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	
    hipMalloc(&vector, N * sizeof(unsigned));
    hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
//    printf("nblocks = %d\n", nblocks);

//here we run  the kernel in a loop which runs 1024 times. 
    for(int i=0;i<N;i++)
        dkernel<<<nblocks, BLOCKSIZE>>>(vector, N,i);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }
    return 0;
}
