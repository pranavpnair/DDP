#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id<vectorsize)   
        vector[id]++;  
}

#define BLOCKSIZE 1024


int main(int nn, char *str[]) {
    unsigned long long N = 1024*1024;
    unsigned *vector, *hvector;
    unsigned vec[N];
    for (int i = 0; i < N; i++) {
		vec[i] = i;
	}
	
    hipMalloc(&vector, N * sizeof(unsigned));
    hipMemcpy(vector, vec, N * sizeof(unsigned), hipMemcpyHostToDevice);
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
//    printf("nblocks = %d\n", nblocks);
    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned ii = 0; ii < N; ++ii) {
    printf("%4d ", hvector[ii]);
    }
    return 0;
}
